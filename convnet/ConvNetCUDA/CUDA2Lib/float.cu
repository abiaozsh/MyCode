#include "hip/hip_runtime.h"



extern "C" __declspec(dllexport) float* allocDeviceMemory(int size)
{
	float* mem;

	hipError_t cudaStatus = hipMalloc((void**)&mem, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		return 0;
	}
	return mem;
}


extern "C" __declspec(dllexport) void freeDeviceMemory(float* point)
{
	hipFree(point);
}

extern "C" __declspec(dllexport) int memcpyHostToDevice(float* hostPoint, float* devicePoint, unsigned int size)
{
	return hipMemcpy(devicePoint, hostPoint, size * sizeof(float), hipMemcpyHostToDevice);
}

extern "C" __declspec(dllexport) int memcpyDeviceToHost(float* devicePoint, float* hostPoint, unsigned int size)
{
	return hipMemcpy(hostPoint, devicePoint, size * sizeof(float), hipMemcpyDeviceToHost);
}

