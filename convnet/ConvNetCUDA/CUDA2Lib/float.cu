
#include "hip/hip_runtime.h"


void* aligned_malloc(size_t size, size_t alignment)
{

	if (alignment < sizeof(void*))
	{
		alignment = sizeof(void*);
	}

	// allocate extra memory and convert to size_t to perform calculations
	char* orig = new char[size + alignment + sizeof(void*)];
	// calculate an aligned position in the allocated region
	// assumption: (size_t)orig does not lose lower bits
	char* aligned =
		orig + (
		(((size_t)orig + alignment + sizeof(void*)) & ~(alignment - 1)) -
		(size_t)orig
		);
	// save the original pointer to use it in aligned_free
	*((char**)aligned - 1) = orig;
	return aligned;
}


void aligned_free(void *aligned)
{
	if (!aligned)return; // behaves as delete: calling with 0 is NOP
	delete[] * ((char**)aligned - 1);
}


size_t zeroCopySizeAlignment(size_t requiredSize)
{
	// Please refer to Intel Zero Copy Tutorial and OpenCL Performance Guide
	// The following statement rounds requiredSize up to the next 64-byte boundary
	return requiredSize + (~requiredSize + 1) % 64;   // or even shorter: requiredSize + (-requiredSize) % 64
}


extern "C" __declspec(dllexport) float* allocfloat(int size)
{
	//extern __m256  __cdecl _mm256_load_ps(float const *);
	//extern void    __cdecl _mm256_store_ps(float *, __m256);

	//cl_float* p_input = NULL;

	int dev_alignment = 64;// zeroCopyPtrAlignment(oclobjects.device);
	//cl_uint dev_alignment = 256;// zeroCopyPtrAlignment(oclobjects.device);
	size_t aligned_size = zeroCopySizeAlignment(sizeof(float)* size);

	float* p_input = (float*)aligned_malloc(aligned_size, dev_alignment);

	return p_input;
}

extern "C" __declspec(dllexport) void freefloat(float* point)
{
	aligned_free(point);
}


extern "C" __declspec(dllexport) float* allocDeviceMemory(int size)
{
	float* mem;

	hipError_t cudaStatus = hipMalloc((void**)&mem, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		return 0;
	}
	return mem;
}


extern "C" __declspec(dllexport) void freeDeviceMemory(float* point)
{
	hipFree(point);
}

extern "C" __declspec(dllexport) int memcpyHostToDevice(float* hostPoint, float* devicePoint, unsigned int size)
{
	return hipMemcpy(devicePoint, hostPoint, size * sizeof(float), hipMemcpyHostToDevice);
}

extern "C" __declspec(dllexport) int memcpyDeviceToHost(float* devicePoint, float* hostPoint, unsigned int size)
{
	return hipMemcpy(hostPoint, devicePoint, size * sizeof(float), hipMemcpyDeviceToHost);
}

