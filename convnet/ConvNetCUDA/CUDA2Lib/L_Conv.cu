#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#define blockSize 1

__global__ void K_CVFWD(
	int filterSize,
	int stride,
	int pad,
	int sx,
	int sy,
	int in_sx,
	int in_sy,
	int in_depth,
	int out_sx,
	int out_sy,
	int out_depth,
	const float* p_filters_w,
	const float* p_in_act_w,
	const float* p_bias_w,
	float* p_out_act_w
	){
	int out_x = blockDim.x * blockIdx.x + threadIdx.x;
	int out_y = blockDim.y * blockIdx.y + threadIdx.y;

	int frame_x = out_x * stride - pad;
	int frame_y = out_y * stride - pad;
	int out_act_sx_out_y = out_sx * out_y;
	// convolve centered at this particular location
	for (int d = 0; d < out_depth; d++)
	{
		//Vol f = this.filters[d];
		int filterIdx = d * filterSize;
		float a = 0.0f;
		for (int fy = 0; fy < sy; fy++)
		{
			int oy = frame_y + fy; // coordinates in the original input array coordinates
			int oy2 = (in_sx * oy);
			int fy2 = (fy * sx);
			for (int fx = 0; fx < sx; fx++)
			{
				int ox = frame_x + fx;
				if (oy >= 0 && oy < in_sy && ox >= 0 && ox < in_sx)
				{
					int fidx = (fy2 + fx) * in_depth + filterIdx;
					int Vidx = (oy2 + ox) * in_depth;
					for (int fd = 0; fd < in_depth; fd++)
					{
						// avoid function call overhead (x2) for efficiency, compromise modularity :(
						a += p_filters_w[fidx + fd] * p_in_act_w[Vidx + fd];
					}
				}
			}
		}
		a += p_bias_w[d];
		p_out_act_w[(out_act_sx_out_y + out_x) * out_depth + d] = a;
	}

}
extern "C" __declspec(dllexport) void CUDA_CVFWD(
	int stride,
	int pad,
	int sx,
	int sy,
	int in_sx,
	int in_sy,
	int in_depth,
	int out_sx,
	int out_sy,
	int out_depth,
	const float* p_filters_w,
	const float* p_in_act_w,
	const float* p_bias_w,
	float* p_out_act_w
	)
{
	int filterSize = sx * sy * in_depth;

	dim3 threads(blockSize, blockSize);
	dim3 grid(out_sx / blockSize, out_sy / blockSize);

	K_CVFWD << < grid, threads >> >(
		filterSize,
		stride,
		pad,
		sx,
		sy,
		in_sx,
		in_sy,
		in_depth,
		out_sx,
		out_sy,
		out_depth,
		p_filters_w,
		p_in_act_w,
		p_bias_w,
		p_out_act_w

		);

	//for (int out_y = 0; out_y < out_sy; out_y++)
	//{
	//	for (int out_x = 0; out_x < out_sx; out_x++)
	//	{
	//	}
	//}
}
