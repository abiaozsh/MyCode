
#include "hip/hip_runtime.h"


extern "C" __declspec(dllexport) int init(int device)
{
	return hipSetDevice(device);
}