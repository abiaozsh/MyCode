#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define blockSize 1
//#define blockSize 16

__global__ void K_FCFWD(
	int out_depth,
	int num_inputs,
	const float* p_in_act_w,
	const float* p_filters_w,
	const float* p_bias_w,
	float* p_out_act_w
	){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	float a = 0.0f;
	int i_num_inputs = i * num_inputs;
	for (int d = 0; d < num_inputs; d++)
	{
		a += p_in_act_w[d] * p_filters_w[i_num_inputs + d];
	}
	a += p_bias_w[i];
	p_out_act_w[i] = a;
}
/*
for (int i = 0; i < out_depth; i++)
{
float a = 0.0f;
int i_num_inputs = i * num_inputs;
for (int d = 0; d < num_inputs; d++)
{
a += p_in_act_w[d] * p_filters_w[i_num_inputs + d]; // for efficiency use Vols directly for now
}
a += p_bias_w[i];
p_out_act_w[i] = a;
}

*/
extern "C" __declspec(dllexport) int CUDA_FCFWD(
	int out_depth,
	int num_inputs,
	const float* p_in_act_w,
	const float* p_filters_w,
	const float* p_bias_w,
	float* p_out_act_w
	){

	K_FCFWD << <out_depth / blockSize, blockSize >> >(
		out_depth,
		num_inputs,
		p_in_act_w,
		p_filters_w,
		p_bias_w,
		p_out_act_w
		);

	return hipGetLastError();

}




__global__ void K_FCBWD(
	int out_depth,
	int num_inputs,

	const float* p_in_act_w,
	const float* p_filters_w,
	const float* p_out_act_dw,

	float* p_in_act_dw,
	float* p_filters_dw,
	float* p_bias_dw
	){
	int d = blockDim.x * blockIdx.x + threadIdx.x;
	
	p_in_act_dw[d] = 0;
	int i_num_inputs = 0;
	float a = 0.0f;
	for (int i = 0; i < out_depth; i++)
	{
		float chain_grad = p_out_act_dw[i];
		a += p_filters_w[i_num_inputs + d] * chain_grad; // grad wrt input data
		//}
		//for (int d = 0; d < num_inputs; d++)
		//{
		p_filters_dw[i_num_inputs + d] += p_in_act_w[d] * chain_grad; // grad wrt params

		i_num_inputs += num_inputs;
	}
	p_in_act_dw[d] = a;
}

__global__ void K_FCBWD2(
	int out_depth,
	int num_inputs,

	const float* p_out_act_dw,

	float* p_bias_dw
	){

	int i_num_inputs = 0;
	for (int i = 0; i < out_depth; i++)
	{
		float chain_grad = p_out_act_dw[i];
		p_bias_dw[i] += chain_grad;
		i_num_inputs += num_inputs;
	}
}

extern "C" __declspec(dllexport) int CUDA_FCBWD(
	int out_depth,
	int num_inputs,

	const float* p_in_act_w,
	const float* p_filters_w,
	const float* p_out_act_dw,

	float* p_in_act_dw,
	float* p_filters_dw,
	float* p_bias_dw
	){


	K_FCBWD << <out_depth / blockSize, blockSize >> >(
		out_depth,
		num_inputs,

		p_in_act_w,
		p_filters_w,
		p_out_act_dw,

		p_in_act_dw,
		p_filters_dw,
		p_bias_dw
		);

	hipError_t err = hipGetLastError();
	if (err != 0)return err;

	K_FCBWD2 << <1, 1 >> >(
		out_depth,
		num_inputs,

		p_out_act_dw,

		p_bias_dw
		);

	return hipGetLastError();

}
