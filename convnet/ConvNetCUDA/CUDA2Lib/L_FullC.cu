#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void K_FCFWD(
	int out_depth,
	int num_inputs,
	const float* p_in_act_w,
	const float* p_filters_w,
	const float* p_bias_w,
	float* p_out_act_w
){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	float a = 0.0f;
	int i_num_inputs = i * num_inputs;
	for (int d = 0; d < num_inputs; d++)
	{
		a += p_in_act_w[d] * p_filters_w[i_num_inputs + d];
	}
	a += p_bias_w[i];
	p_out_act_w[i] = a;
}

extern "C" __declspec(dllexport) int CUDA_FCFWD(
	int out_depth,
	int num_inputs,
	const float* p_in_act_w,
	const float* p_filters_w,
	const float* p_bias_w,
	float* p_out_act_w
	){

	K_FCFWD << <out_depth/1,1 >> >(
		out_depth,
		num_inputs,
		p_in_act_w,
		p_filters_w,
		p_bias_w,
		p_out_act_w
	);

	return hipGetLastError();

}
